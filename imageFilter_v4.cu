#include "hip/hip_runtime.h"

// Based on CUDA SDK template from NVIDIA

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <unistd.h>

// includes, project
#include <cutil_inline.h>

#define BLOCK_DIM_X 16
#define BLOCK_DIM_Y 16

#define max(a,b) (((a)>(b))?(a):(b))
#define min(a,b) (((a)<(b))?(a):(b))

// loads filter coefficients from file fname,
// allocates memory through parray and stores width and height of filter through pwidth and pheight
int loadFilter(char* fname, float** parray, unsigned int *pwidth, unsigned int *pheight)
{
    FILE* fp;

    if( (fp=fopen(fname, "r")) == NULL)
    {
        fprintf(stderr,"Failed to open filter file %s\n",fname);
        return -1;
    }

    if(fscanf(fp,"%u %u",pwidth,pheight)!=2) {
        fprintf(stderr,"Failed to read header of filter file %s\n",fname);
        return -1;
    }

    *parray = (float *) malloc((*pwidth)*(*pheight)*sizeof(float));

    int i;
    for(i=0;i<(*pwidth)*(*pheight);i++)
    {
        if(fscanf(fp,"%f",(*parray+i))!=1) {
            fprintf(stderr,"Failed to read data of filter file %s\n",fname);
            return -1;
        }
    }

    fclose(fp);

    return 0;
}


// filter code to run on the host
void filterHost(unsigned int *h_idata, unsigned int w, unsigned int h,
        float* filter, unsigned int fw, unsigned int fh,
        unsigned int* reference)
{
    int i,j,k,l;

    int fh_2 = fh/2;
    int fw_2 = fw/2;

    for(i=0; i<h; i++) //height image
    {
        for(j=0; j<w; j++) //width image
        {
            float sum = 0;
            for(k=-fh_2; k<=fh_2; k++) //filter height
            {
                for(l=-fw_2; l<=fw_2; l++) //filter width
                {
                    if( (i+k >= 0) && (i+k < h))
                        if( (j+l >=0) && (j+l < w)) {
                            sum += h_idata[(i+k)*w + j+l]*filter[(k+fh/2)*fw + l+fw/2];
                        }

                }
            }
            reference[i*w+j] = min(max(sum,0),255);
        }
    }
}

__global__ void renderFilteredImage(unsigned int *idata, unsigned int w, unsigned int h,
        float *filter, unsigned int fw, unsigned int fh,
        unsigned int *odata) {

    int i, j, k, l;
    unsigned int fw_2, fh_2;
    float sum;

    fw_2 = fw/2;
    fh_2 = fh/2;
    sum = 0;

    // convalescence
    j = threadIdx.x;
    i = blockIdx.y;

    for (k =- fh_2; k <= fh_2; k++) //filter height
    {
        for (l =- fw_2; l <= fw_2; l++) //filter width
            if( (i+k >= 0) && (i+k < h))
                if( (j+l >=0) && (j+l < w))
                    sum += idata[(i+k)*w + j+l] * filter[(k+fh/2)*fw + l+fw/2];

        odata[i*w+j] = min(max(sum,0),255);
    }
}

void filterDevice(unsigned int *h_idata, unsigned int w, unsigned int h,
        float* filter, unsigned int fw, unsigned int fh,
        unsigned int* h_odata)
{
    unsigned int *idata, *odata, data_size, filter_size;
    float *f;

    data_size = w * h * sizeof(unsigned int);
    filter_size = fw * fh * sizeof(float);

    dim3 dimGrid(w);
    dim3 dimBlock(h);

    // memory allocation
    hipMalloc((void **)&idata, data_size);
    hipMalloc((void **)&f, filter_size);
    hipMalloc((void **)&odata, data_size);

    // copy image and filter to device (CPU->GPU)
    hipMemcpy(idata, h_idata, data_size, hipMemcpyHostToDevice);
    hipMemcpy(f, filter, filter_size, hipMemcpyHostToDevice);

    // render filtered image on GPU
    renderFilteredImage<<<dimGrid, dimBlock>>>(idata, w, h, f, fw, fh, odata);

    // copy result from device to host (GPU->CPU)
    hipMemcpy(h_odata, odata, data_size, hipMemcpyDeviceToHost);

    // free allocated memory
    hipFree(idata);
    hipFree(odata);
    hipFree(f);
}

// print command line format
void usage(char *command)
{
    printf("Usage: %s [-h] [-d device] [-i inputfile] [-o outputfile] [-f filterfile]\n",command);
}

// main
int main( int argc, char** argv)
{

    // default command line options
    int deviceId = 0;
    char *fileIn="lena.pgm",*fileOut="lenaOut.pgm",*fileFilter="filter.txt";

    // parse command line arguments
    int opt;
    while( (opt = getopt(argc,argv,"d:i:o:f:h")) !=-1)
    {
        switch(opt)
        {

            case 'd':
                if(sscanf(optarg,"%d",&deviceId)!=1)
                {
                    usage(argv[0]);
                    exit(1);
                }
                break;

            case 'i':
                if(strlen(optarg)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }

                fileIn = strdup(optarg);
                break;
            case 'o':
                if(strlen(optarg)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }
                fileOut = strdup(optarg);
                break;
            case 'f':
                if(strlen(optarg)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }
                fileFilter = strdup(optarg);
                break;
            case 'h':
                usage(argv[0]);
                exit(0);
                break;

        }
    }

    // select cuda device
    cutilSafeCall( hipSetDevice( deviceId ) );

    // create events to measure host filter time and device filter time
    hipEvent_t startH, stopH, startD, stopD;
    hipEventCreate(&startH);
    hipEventCreate(&stopH);
    hipEventCreate(&startD);
    hipEventCreate(&stopD);


    // allocate host memory
    unsigned int* h_idata=NULL;
    unsigned int h,w;
    //load pgm
    if (cutLoadPGMi(fileIn, &h_idata, &w, &h) != CUTTrue) {
        printf("Failed to load image file: %s\n", fileIn);
        exit(1);
    }

    //load filter
    float *filter;
    unsigned int fh, fw;
    if(loadFilter(fileFilter, &filter, &fw, &fh)==-1)
    {
        printf("Failed to load filter file: %s\n",fileFilter);
        exit(1);
    }

    // allocate mem for the result on host side
    unsigned int* h_odata = (unsigned int*) malloc( h*w*sizeof(unsigned int));
    unsigned int* reference = (unsigned int*) malloc( h*w*sizeof(unsigned int));

    // filter at host
    hipEventRecord( startH, 0 );
    filterHost(h_idata, w, h, filter, fw, fh, reference);
    hipEventRecord( stopH, 0 );
    hipEventSynchronize( stopH );

    // filter at GPU
    hipEventRecord( startD, 0 );
    filterDevice(h_idata, w, h, filter, fw, fh, h_odata);
    hipEventRecord( stopD, 0 );
    hipEventSynchronize( stopD );

    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");

    float timeH, timeD;
    hipEventElapsedTime( &timeH, startH, stopH );
    printf( "Host processing time: %f (ms)\n", timeH);
    hipEventElapsedTime( &timeD, startD, stopD );
    printf( "Device processing time: %f (ms)\n", timeD);

    // save output image
    if (cutSavePGMi(fileOut, reference, w, h) != CUTTrue) {
        printf("Failed to save image file: %s\n", fileOut);
        exit(1);
    }

    // cleanup memory
    cutFree( h_idata);
    free( h_odata);
    free( reference);
    free( filter);

    cutilDeviceReset();
}
